#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "io.h"
#include "rand.h"
#include <omp.h>
#include <hip/hip_runtime.h>

/* batch size */
#define B 8
/* input size */
#define X 784
/* hidden size */
#define H 1000
/* output size */
#define Y 10

#define NUM_THREADS 4
#define ITERATIONS 1000000
#define TARGET_ACC 0.95f
#define STATS_INTERVAL 50000
#define SMOOTHING 0.99999f
#define LEARNING_RATE 5 * 1e-4f
#define DATAPOINTS 50000
#define WEIGHT_DECAY .0f
#define DROPOUT 0.0
#define LOGISTIC 0
#define RELU 1
#define TANH 0
// #define RHO_TARGET 0.2f
// #define BETA 1e-5f

extern void randn(float *out, float mean, float std, int n);

unsigned char inputs[X * DATAPOINTS];
unsigned char labels[DATAPOINTS];

double get_time()
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec + tv.tv_usec * 1e-6);
}

void cuda_check_error(hipError_t err)
{
  if (err != hipSuccess)
  {
    printf("CUDA error (%d): %s  \n", err, hipGetErrorString(err));
    exit(-1);
  }
}

__global__ void myKernel(float *dy, float *dv, float *dh, float *p, float *t, float *h, float *v) {
    int b = blockIdx.x;
    int j = threadIdx.x;
    int k = threadIdx.y;
    dy[b * Y + k] = p[b * Y + k] - t[b * Y + k];
    dv[k * H + j] += h[b * H + j] * dy[b * Y + k];
    dh[b * H + j] += v[k * H + j] * dy[b * Y + k];
}

int main(int argc, char **argv)
{
  /* command line argument */
  if (argc > 1)
    if (0 == strcmp(argv[1], "help"))
    {
      printf("usage: %s max_iters lr decay\n", argv[0]);
      return 0;
    }

  omp_set_num_threads(NUM_THREADS);


  // float *dy, float *dv, float *dh, float *p, float *t, float *h, float *v
  /* x -w-> h -v-> y */
  float *x, *h_h, *d_h, *y, *h_p, *d_p, *h_t, *d_t, *c; /*states*/
  float *w, *h_v, *d_v;                 /*weights*/
  float *h_dh, *d_dh, *h_dy, *d_dy;               /*states-grads*/
  float *dw, *h_dv, *d_dv;               /*weight-grads*/
  float *m;                     /*dropout*/

  /* allocate memory for arrays */
  x = (float *)malloc(sizeof(float) * X * B);
  w = (float *)malloc(sizeof(float) * X * H);
  dw = (float *)malloc(sizeof(float) * X * H);
  h_h = (float *)malloc(sizeof(float) * H * B);
  h_dh = (float *)malloc(sizeof(float) * H * B);
  m = (float *)malloc(sizeof(float) * H * B);
  h_v = (float *)malloc(sizeof(float) * H * Y);
  h_dv = (float *)malloc(sizeof(float) * Y * H);
  h_dy = (float *)malloc(sizeof(float) * Y * B);
  y = (float *)malloc(sizeof(float) * Y * B);
  h_p = (float *)malloc(sizeof(float) * Y * B);
  c = (float *)malloc(sizeof(float) * Y * B);
  h_t = (float *)malloc(sizeof(float) * Y * B);

  cuda_check_error(hipMalloc((void **)&d_h, sizeof(float) * H * B));
  cuda_check_error(hipMalloc((void **)&d_dh, sizeof(float) * H * B));
  cuda_check_error(hipMalloc((void **)&d_v, sizeof(float) * H * Y));
  cuda_check_error(hipMalloc((void **)&d_dv, sizeof(float) * Y * H));
  cuda_check_error(hipMalloc((void **)&d_dy, sizeof(float) * Y * B));
  cuda_check_error(hipMalloc((void **)&d_p, sizeof(float) * Y * B));
  cuda_check_error(hipMalloc((void **)&d_t, sizeof(float) * Y * B));

  dim3 threadsPerBlock(H,Y);
  dim3 numBlocks(B);

  /* init stats */
  float smooth_act = 0.0f;
  float smooth_ce = logf(Y);
  float smooth_acc = 1.0f / Y;

  /* set default values for command line arguments */
  int max_iters = argc > 1 ? atoi(argv[1]) : ITERATIONS;
  float lr = argc > 1 ? atof(argv[2]) : LEARNING_RATE;
  float decay = argc > 1 ? atof(argv[3]) : WEIGHT_DECAY;

  /* load data */
  if (0 > load("../data/train-images-idx3-ubyte",
               16, X * DATAPOINTS, inputs))
    return -1;
  if (0 > load("../data/train-labels-idx1-ubyte",
               8, DATAPOINTS, labels))
    return -1;

  /* init weights */
  randn(w, .0f, 0.1f, X * H);
  randn(v, .0f, 0.1f, Y * H);

  double gflops_per_sample =
      (double)(2 * (X * H + H * Y) * 2) /
      (1 << 30);

  int samples = 0, iters = 0;
  srand(33);

  double t0 = get_time();
  double start_time = t0;

  /* */
  do
  {
    /* random sample */
    int r[B];

    for (int b = 0; b < B; b++)
      r[b] = random() % DATAPOINTS;

    memset(t, 0, sizeof(float) * Y * B);
    memset(h, 0, sizeof(float) * H * B);
    memset(y, 0, sizeof(float) * Y * B);

    for (int b = 0; b < B; b++)
    {
      t[b * Y + labels[r[b]]] = 1.0f;
      for (int i = 0; i < X; i++)
        x[b * X + i] = inputs[r[b] * X + i] / 255.0f;
    }

    /* h := w'x */
    /* col major */
    /* h [H rows, B cols] */
    /* w [X rows, H cols] */
    /* x [X rows, B cols] */
    #pragma omp parallel for// collapse(3)
    for (int j = 0; j < H; j++)
      for (int i = 0; i < X; i++)
        for (int b = 0; b < B; b++)
          h[b * H + j] +=
              w[j * X + i] * x[b * X + i];

    /* nonlinearity */
    for (int j = 0; j < H * B; j++)
#if LOGISTIC
      h[j] = 1.0f / (1.0f + expf(-h[j]));
#endif
#if RELU
    h[j] = h[j] < 0.0f ? 0.0f : h[j];
#endif
#if TANH
    h[j] = tanhf(h[j]);
#endif

    if (DROPOUT > 0)
    {
      for (int j = 0; j < H * B; j++)
      {
        m[j] = ((float)random() / (float)RAND_MAX) < DROPOUT ? 0.0f : 1.0f;
        h[j] *= m[j];
      }
    }

    float act_sum = 0.0f;
    for (int i = 0; i < H * B; i++)
      act_sum += h[i];

    smooth_act = SMOOTHING * smooth_act + (1.0f - SMOOTHING) * act_sum / (H * B);

    /* y := vh */
    /* col major */
    for (int b = 0; b < B; b++)
      for (int j = 0; j < H; j++)
        for (int k = 0; k < Y; k++)
          y[b * Y + k] += v[k * H + j] * h[b * H + j];

    /* p := softmax(y) */
    for (int b = 0; b < B; b++)
    {
      float m0 = .0f; /* find max */
      for (int k = 0; k < Y; k++)
        m0 = k > 0 && (y[b * Y + k] > m0) ? y[b * Y + k] : m0;

      float sum = .0f;

      for (int k = 0; k < Y; k++)
      {
        p[b * Y + k] = expf(y[b * Y + k] - m0);
        sum += p[b * Y + k];
      }

      for (int k = 0; k < Y; k++)
        p[b * Y + k] /= sum;
    }

    /* forward pass end */
    /* bookkeeping for stats */
    int argmax[B];
    float probmax[B];
    for (int b = 0; b < B; b++)
    {
      argmax[b] = -1;
      probmax[b] = .0f;
      for (int k = 0; k < Y; k++)
      {
        if (probmax[b] < p[b * Y + k] || k == 0)
        {
          probmax[b] = p[b * Y + k];
          argmax[b] = k;
        }
        c[b * Y + k] = -logf(p[b * Y + k]) * t[b * Y + k];
        smooth_ce = smooth_ce * SMOOTHING +
                    (1.0f - SMOOTHING) * c[b * Y + k];
      }
      smooth_acc = smooth_acc * SMOOTHING +
                   (1.0f - SMOOTHING) * (argmax[b] == labels[r[b]]);
    }

    if (0 == (samples % STATS_INTERVAL) &&
        samples > 0)
    {
      float time_d = get_time() - t0;
      float samples_per_sec = STATS_INTERVAL / time_d;
      float gflops_per_sec = samples_per_sec *
                             gflops_per_sample;
      printf("[%4.3f s] "
             "acc=%3.2f%%, "
             "ce=%3.3f, "
             "%.2f samples/sec, "
             "%.2f gflop/s\n",
             get_time() - start_time, 100.0 * smooth_acc, smooth_ce,
             samples_per_sec, gflops_per_sec);

      t0 = get_time();
    }

    /* backprop begin */
    /* reset grads */
    memset(dh, 0, sizeof(float) * H * B);
    memset(dw, 0, sizeof(float) * H * X);
    memset(dv, 0, sizeof(float) * H * Y);

    /* dy */
    //for (int b = 0; b < B; b++)
    //  for (int k = 0; k < Y; k++)
    //    dy[b * Y + k] = p[b * Y + k] - t[b * Y + k];

    /* dv := h * dy' */
    //for (int b = 0; b < B; b++)
    //  for (int j = 0; j < H; j++)
    //    for (int k = 0; k < Y; k++)
    //      dv[k * H + j] += h[b * H + j] * dy[b * Y + k];

    /* dh := v * dy */
    //for (int b = 0; b < B; b++)
    //  for (int j = 0; j < H; j++)
    //    for (int k = 0; k < Y; k++)
    //      dh[b * H + j] += v[k * H + j] * dy[b * Y + k];

    //#pragma omp parallel for
    cuda_check_error(hipMalloc((void **)&d_h, sizeof(float) * H * B));
    cuda_check_error(hipMalloc((void **)&d_dh, sizeof(float) * H * B));
    cuda_check_error(hipMalloc((void **)&d_v, sizeof(float) * H * Y));
    cuda_check_error(hipMalloc((void **)&d_dv, sizeof(float) * Y * H));
    cuda_check_error(hipMalloc((void **)&d_dy, sizeof(float) * Y * B));
    cuda_check_error(hipMalloc((void **)&d_p, sizeof(float) * Y * B));
    cuda_check_error(hipMalloc((void **)&d_t, sizeof(float) * Y * B));


    // Copy the memory from the host to the device
    cuda_check_error(hipMemcpy(d_h, h_h, sizeof(float) * H * B, hipMemcpyHostToDevice));
    cuda_check_error(hipMemcpy(d_dh, h_dh, sizeof(float) * H * B, hipMemcpyHostToDevice));
    cuda_check_error(hipMemcpy(d_v, h_v, sizeof(float) * H * Y, hipMemcpyHostToDevice));
    cuda_check_error(hipMemcpy(d_dv, h_dv, sizeof(float) * Y * H, hipMemcpyHostToDevice));
    cuda_check_error(hipMemcpy(d_p, h_p, sizeof(float) * Y * B, hipMemcpyHostToDevice));
    cuda_check_error(hipMemcpy(d_t, h_t, sizeof(float) * Y * B, hipMemcpyHostToDevice));

    // Launch the kernel
    myKernel<<<numBlocks, threadsPerBlock>>>(d_dy, d_dv, d_dh, d_p, d_t, d_h, d_v);

    // Copy the memory from the device to the host
    cuda_check_error(hipMemcpy(h_dy, d_dy, sizeof(float) * Y * B, hipMemcpyDeviceToHost));
    cuda_check_error(hipMemcpy(h_dv, d_dv, sizeof(float) * Y * H, hipMemcpyDeviceToHost));
    cuda_check_error(hipMemcpy(h_dh, d_dh, sizeof(float) * H * B, hipMemcpyDeviceToHost));

    /*
    for (int b = 0; b < B; b++) {
        for (int j = 0; j < H; j++) {
            for (int k = 0; k < Y; k++) {
                dy[b * Y + k] = p[b * Y + k] - t[b * Y + k];
                dv[k * H + j] += h[b * H + j] * dy[b * Y + k];
                dh[b * H + j] += v[k * H + j] * dy[b * Y + k];
            }
        }
    }
    */

    /* nonlinearity on h */
    //#pragma omp parallel for
    for (int j = 0; j < H * B; j++)
#if LOGISTIC
      dh[j] = dh[j] * h[j] * (1.0f - h[j]);
#endif
#if RELU
    dh[j] = dh[j] * h[j];
#endif
#if TANH
    dh[j] = dh[j] * (1.0f - h[j] * h[j]);
#endif

    /* dw := x * dh' */
    #pragma omp parallel for
    for (int j = 0; j < H; j++) {
        for (int i = 0; i < X; i++) {
            for (int b = 0; b < B; b++) {
                dw[j * X + i] += x[b * X + i] * dh[b * H + j];
            }
        }
    }
    /* backprop end */

    /* adjust weights */
    //#pragma omp parallel for
    for (int i = 0; i < H * X; i++) {
        w[i] = w[i] * (1.0f - decay) - dw[i] * lr;
    }
    //#pragma omp parallel for
    for (int i = 0; i < H * Y; i++) {
        v[i] = v[i] * (1.0f - decay) - dv[i] * lr;
    }

    samples += B;

  } while (iters++ < max_iters && smooth_acc < TARGET_ACC);

  /* cleanup */
  free(x), free(w), free(dw);
  free(h), free(dh);
  free(m);
  free(v), free(dv);
  free(y), free(dy);
  free(p), free(c), free(t);

  cuda_check_error(hipFree(d_h));
  cuda_check_error(hipFree(d_dh));
  cuda_check_error(hipFree(d_v));
  cuda_check_error(hipFree(d_dv));
  cuda_check_error(hipFree(d_dy));
  cuda_check_error(hipFree(d_p));
  cuda_check_error(hipFree(d_t));



  return 0;
}
